#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "helper.h"

__global__ void calcTrigAngles(const CityPos *posArray, CityTrigFunc *trigFuncArray) {
    // compute global element index
  	unsigned int globalID = threadIdx.x + blockDim.x * blockIdx.x;

    CityPos pos = posArray[globalID];
    CityTrigFunc trigFunc;

    // compute sin and cos of phi and of theta for the city with id globalID
    double phi = (90.f - pos.lat) * DEGREE_TO_RADIANS;
    double theta = pos.lon * DEGREE_TO_RADIANS;
    trigFunc.sinPhi = sin(phi);
    trigFunc.cosPhi = cos(phi);
    trigFunc.sinTheta = sin(theta);
    trigFunc.cosTheta = cos(theta);
    trigFuncArray[globalID] = trigFunc;
}

__global__ void calcAccesiblePopulations(const CityTrigFunc *trigFuncArray, const int *popArrayIn,
                                        int *popArrayOut, const size_t numberOfCities, double kmRange) {
  	// compute global element index
  	unsigned int globalID = threadIdx.x + blockDim.x * blockIdx.x;

  	// avoid out of bounds exceptions
  	if (globalID < numberOfCities) {
        // take sin and cos for phi and theta for the city assigned to the current thread
        CityTrigFunc trigFunc1 = trigFuncArray[globalID];
        double sinPhi1 = trigFunc1.sinPhi;
        double cosPhi1 = trigFunc1.cosPhi;
        double sinTheta1 = trigFunc1.sinTheta;
        double cosTheta1 = trigFunc1.cosTheta;

        // in totalPop I would store the accessible population of the city
        // assigned to the current thread
        int totalPop = 0;
        for (size_t cityID = 0; cityID < numberOfCities; cityID++) {
            // take sin and cos for phi and theta for the current city in the loop
            CityTrigFunc trigFunc2 = trigFuncArray[cityID];
            double sinPhi2 = trigFunc2.sinPhi;
            double cosPhi2 = trigFunc2.cosPhi;
            double sinTheta2 = trigFunc2.sinTheta;
            double cosTheta2 = trigFunc2.cosTheta;

            // calculate cs
            double cosTheta1MinusTheta2 = cosTheta1 * cosTheta2 + sinTheta1 * sinTheta2;
            double cs = sinPhi1 * sinPhi2 * cosTheta1MinusTheta2 + cosPhi1 * cosPhi2;
            if (cs > 1) {
                cs = 1;
            } else if (cs < -1) {
                cs = -1;
            }

            double negate = double(cs < 0);
            if (cs < 0) {
                cs = -cs;
            }

            // calculate acos(cs) using Nvidia implementation
            double aCosOfCs = -0.0187293;
            aCosOfCs = aCosOfCs * cs;
            aCosOfCs = aCosOfCs + 0.0742610;
            aCosOfCs = aCosOfCs * cs;
            aCosOfCs = aCosOfCs - 0.2121144;
            aCosOfCs = aCosOfCs * cs;
            aCosOfCs = aCosOfCs + 1.5707288;
            aCosOfCs = aCosOfCs * sqrt(1.0-cs);
            aCosOfCs = aCosOfCs - 2 * negate * aCosOfCs;
            aCosOfCs = negate * 3.14159265358979 + aCosOfCs;

            // calculate distance
            double dist = 6371.f * aCosOfCs;
            if (dist <= kmRange) {
                totalPop += popArrayIn[cityID];
            }
        }

        popArrayOut[globalID] = totalPop;
  	}
}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    // declare host arrays
    CityPos *hostPosArray;
    int *hostPopArray;

    // declare device arrays
    CityPos *devicePosArray;
    CityTrigFunc *deviceTrigFuncArray;
    int *devicePopArrayIn;
    int *devicePopArrayOut;

    // constants
    const size_t blockSize = 256;

    for(int argcID = 1; argcID < argc; argcID += 3) {
        double kmRange = atof(argv[argcID]);
        size_t numberOfCities = getNumberOfLines(argv[argcID + 1]);

        // allocate host arrays for the individual file
        hostPosArray = (CityPos *) malloc(numberOfCities * sizeof(CityPos));
        hostPopArray = (int *) malloc(numberOfCities * sizeof(int));

        // allocate device arrays for the individual file
        hipMalloc((void **) &devicePosArray, numberOfCities * sizeof(CityPos));
        hipMalloc((void **) &devicePopArrayIn, numberOfCities * sizeof(int));
        hipMalloc((void **) &devicePopArrayOut, numberOfCities * sizeof(int));
        hipMalloc((void **) &deviceTrigFuncArray, numberOfCities * sizeof(CityTrigFunc));

        // populating the arrays with data
        readFile(argv[argcID + 1], hostPosArray, hostPopArray);

        // copy host arrays to device
        hipMemcpy(devicePosArray, hostPosArray, numberOfCities * sizeof(CityPos), hipMemcpyHostToDevice);
        hipMemcpy(devicePopArrayIn, hostPopArray, numberOfCities * sizeof(int), hipMemcpyHostToDevice);

        // calling kernel functions
        size_t numberOfBlocks = (numberOfCities / blockSize) + 1;
        calcTrigAngles<<<numberOfBlocks, blockSize>>>(devicePosArray, deviceTrigFuncArray);
        calcAccesiblePopulations<<<numberOfBlocks, blockSize>>>(deviceTrigFuncArray, devicePopArrayIn, devicePopArrayOut, numberOfCities, kmRange);

        // copy result back to host
        hipMemcpy(hostPopArray, devicePopArrayOut, numberOfCities * sizeof(int), hipMemcpyDeviceToHost);

        // writing output
        writeFile(argv[argcID + 2], hostPopArray, numberOfCities);

        // free host arrays
        free(hostPosArray);
        free(hostPopArray);

        // free device arrays
        hipFree(devicePosArray);
        hipFree(devicePopArrayIn); 
        hipFree(devicePopArrayOut);
        hipFree(deviceTrigFuncArray);
    }
}
